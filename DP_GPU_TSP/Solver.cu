#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <fstream>
#include "Binary_operations.h"
#include "Problem.h"

#define DropBit(h,z) (h=(((h)&devPattern1[z])>>1) | ((h)&devPattern2[z]))

#define MAX_N 32
#define MAX_COST 1000000

unsigned int * createMap(int N, int &totSize);
unsigned int computeSizeF(int);


__global__ void kernelInit(int n,unsigned int* devPattern1, unsigned int* devPattern2)
{
  int i,j;
  int m = n-1;
  for(i = 0; i < m; i++)
  {  
    devPattern2[i]=(1 << i)-1; 
    j=(1 << m-i-1)-1;  
    devPattern1[i]=j<<(i+1);
  } 
}

//------------------------------------------------------------------------------

__global__ void kernelFirst(
                           int n,                     
                           float *setup,
                           float* f,
                           unsigned int* devPattern1, unsigned int* devPattern2
                           )
{
    int i = blockIdx.x; // city
    int j = threadIdx.x; // subset

    if(i == j)
      return;

    int s = 1<<j;
    DropBit(s,i);

    f[s*n + i] = setup[i*n+j] + setup[j*n+(n-1)];
} 

//------------------------------------------------------------------------------

__global__ void kernelStep(
                           int n,  
                           int nS,                                            
                           float *setup,                
                           float *f,
                           unsigned int* map,
                           int ne,                      // number of '1'
                           unsigned int* devPattern1, 
                           unsigned int* devPattern2
                           )
{
    int i = threadIdx.x;   //city
    unsigned int is =  blockIdx.x * gridDim.y + blockIdx.y ;                   
    is=blockDim.y * is + threadIdx.y;  // index of the set in binary map of sets
    if(is >= nS || i >= n)
      return;

    int N = map[0];
    unsigned int *valMap = map+2+N+1;
    int spB = map[1 + ne] ;
    unsigned int s = valMap[spB + is];  // now f(s,i) is to be computed

    if(GetBit(s,i) == 1)
      return;
    
    float *iSetup = setup + i*n;

    int j; 
    float dmin = MAX_COST;
    // loop over elements in set s
    for(j = 0; j < n-1; j++)
    {
       if(GetBit(s,j) == 0 )
         continue;

       unsigned int curS = s;
       DropBit(curS, j);
       float cj = iSetup[j] + f[curS*n + j];    // corresponds to  a_ij + f(S\j, j)
       if(cj < dmin)
         dmin = cj;
    }

    DropBit(s,i);
    f[s*n+i]=dmin;      
} 

//------------------------------------------------------------------------------

__global__ void kernelGetTour(
                           int n,                                               
                           float *setup,                
                           float *f,
                           int* tour,
                           unsigned int* devPattern1, 
                           unsigned int* devPattern2
                           )
{
  int i, ic, j, k, imin;
  float dmin;
  unsigned int s=(1<<(n-1))-1; // binary string of type 11...1
  
  int m = n-1;
  int c[MAX_N];
  for(i=0;i<m;i++)
  {
    c[i]=i;
  }
  
  float part_cost=0;
 
  int prev = n-1;
  for(k = 0; k < n-1; k++)  
  {    
    dmin=MAX_COST; 
    for(ic = 0; ic < m; ic++)
    {
      i=c[ic];
      j=prev;  // previous city in a tour
      unsigned int curS=s;
      DropBit(curS,i);
      float d = part_cost + setup[j*n + i] + f[curS * n + i];
      if(d<dmin)
      {
        dmin=d;
        imin=ic;
      }
    }
   
    i=c[imin];
    tour[k]=i;
    SetBit0(s,i);
    c[imin] = c[m-1];
    part_cost += setup[prev*n + i];
    prev=i;       
    m--;
  }
  tour[n-1]=n-1;
}

//------------------------------------------------------------------------------

float solve(Problem *p, int *tour) {

	int i, n;
    n = p->n;     

    unsigned int fSize = computeSizeF(n);    
    printf("\nmemory use %iM\n", fSize*sizeof(int)/1024/1024);
    
    float* devF;
    hipMalloc(   (void**)&devF,    fSize * sizeof(float)   );

    int mapSize;
    unsigned int *map = createMap(n-1, mapSize);
    unsigned int* devMap;
    hipMalloc((void**)&devMap, mapSize * sizeof(int) );
    hipMemcpy(devMap,    map,  mapSize * sizeof(int), hipMemcpyHostToDevice);
    
    // distances
    float *s1 = new float [n*n];
    int k=0;
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < n; j++) {
            s1[k] = p->a[i][j];
            k++;
        }
    }

    float *devS1;
    hipMalloc((void**)&devS1, n * n * sizeof(float) );
    hipMemcpy(devS1,    s1,   n * n * sizeof(float), hipMemcpyHostToDevice);
       
    unsigned int *sizeMap = map+2;
	float* F = new float[fSize];
	
	unsigned int* devPattern1;
	unsigned int* devPattern2;
	hipMalloc((void**)&devPattern1, n * sizeof(int) );
	hipMalloc((void**)&devPattern2, n * sizeof(int) );
      
    hipEvent_t start, stop;       
    float gpuTime = 0.0f;    

    hipEventCreate(&start); 
    hipEventCreate(&stop) ; 
    hipEventRecord ( start, 0 ); 
      
    kernelInit<<<1, 1>>>(n,devPattern1,devPattern2);

    // solve for number of ones equal 1
    kernelFirst<<<n-1, n-1>>>(
                            n,
                            devS1,              
                            devF,
                            devPattern1,
                            devPattern2
                          );

    // main cycle: iterate over sets cardinality (number of ones in a binary representation)
    for(int ne = 2; ne <= n-2;  ne++)    
    {
        int nS = sizeMap[ne] - sizeMap[ne-1]; // number of sets of cardinality "ne"  

        int blocksPart = 256;
        int divisor=16;
        int  nThr = nS  /  blocksPart;       
        if(nS % blocksPart > 0)
           nThr++;       
                       
        dim3 blocks  = dim3(blocksPart/divisor, nThr);
        dim3 threads = dim3(n-1, divisor);        

	    kernelStep<<<blocks,threads>>>(
                           n,  
                           nS,                                             
                           devS1,                     
                           devF,
                           devMap,
                           ne,                      // number of '1'                           
                           devPattern1, devPattern2
                           );
         
    } 

    int* devTour;
    hipMalloc((void**)&devTour, n * sizeof(int));

    kernelGetTour<<<1, 1>>>(n, devS1, devF, devTour, devPattern1, devPattern2);
    hipMemcpy(tour, devTour, n * sizeof(int), hipMemcpyDeviceToHost);
    
    float cost=0;
    for(i=0;i<n-1;i++)
    {
      cost += p->a[tour[i]][tour[i+1]];
    }
    cost += p->a[tour[n-1]][tour[0]];
        
    hipEventRecord ( stop, 0 ); 
    hipEventSynchronize (stop) ; 
    hipEventElapsedTime(&gpuTime, start, stop ); 
    printf("time spent executing by the GPU: %.2f  millseconds\n", gpuTime ); 
    hipEventDestroy(start);
    hipEventDestroy(stop); 

  	hipFree(devS1);
    hipFree(devMap);
    hipFree(devF);
    hipFree(devTour);
    hipFree(devPattern1);
    hipFree(devPattern2);

    return cost;
 }
